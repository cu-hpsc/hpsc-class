#include "hip/hip_runtime.h"
__global__ void copy(float *dst, float *src) {
  int iblock = blockIdx.x + blockIdx.y * gridDim.x;
  int index  = threadIdx.x + TILE_SIZE * iblock * blockDim.x;
  float a[TILE_SIZE]; // allocated in registers
  for (int i=0; i<TILE_SIZE; i++)
    a[i] = src[index + i * blockDim.x];
  for (int i=0; i<TILE_SIZE; i++)
    dst[index + i * blockDim.x] = a[i];
}
